
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// these are just for timing measurments
#include <time.h>
// Code that reads values from a 2D grid and for each node in the grid finds the minumum
// value among all values stored in cells sharing that node, and stores the minumum
// value in that node.

// To compile it with nvcc execute: nvcc -O2 -o grid3 grid3.cu
// Modified by Bob Crovella NVIDIA Corp. 12/2011 to demonstrate CUDA

//define the window size (square window) and the data set size
#define WSIZE 16
#define DATAHSIZE 20000
#define DATAWSIZE 14000
#define CHECK_VAL 1
#define MIN(X,Y) ((X<Y)?X:Y)
#define BLKWSIZE 32
#define BLKHSIZE 32

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

typedef int oArray[DATAHSIZE];
typedef int iArray[DATAHSIZE+WSIZE];

__global__ void cmp_win(oArray *output, const iArray *input)
{
    __shared__ int smem[(BLKHSIZE + (WSIZE-1))][(BLKWSIZE + (WSIZE-1))];
    int tempout, i, j;
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    if ((idx < DATAHSIZE+WSIZE) && (idy < DATAWSIZE+WSIZE))
      smem[threadIdx.y][threadIdx.x]=input[idy][idx];
    if ((idx < DATAHSIZE+WSIZE) && (idy < DATAWSIZE) && (threadIdx.y > BLKWSIZE - WSIZE))
      smem[threadIdx.y + (WSIZE-1)][threadIdx.x] = input[idy+(WSIZE-1)][idx];
    if ((idx < DATAHSIZE) && (idy < DATAWSIZE+WSIZE) && (threadIdx.x > BLKHSIZE - WSIZE))
      smem[threadIdx.y][threadIdx.x + (WSIZE-1)] = input[idy][idx+(WSIZE-1)];
    if ((idx < DATAHSIZE) && (idy < DATAWSIZE) && (threadIdx.x > BLKHSIZE - WSIZE) && (threadIdx.y > BLKWSIZE - WSIZE))
      smem[threadIdx.y + (WSIZE-1)][threadIdx.x + (WSIZE-1)] = input[idy+(WSIZE-1)][idx+(WSIZE-1)];
    __syncthreads();
    if ((idx < DATAHSIZE) && (idy < DATAWSIZE)){
      tempout = output[idy][idx];
      for (i=0; i<WSIZE; i++)
        for (j=0; j<WSIZE; j++)
          if (smem[threadIdx.y + i][threadIdx.x + j] < tempout)
            tempout = smem[threadIdx.y + i][threadIdx.x + j];
      output[idy][idx] = tempout;
      }
}

int main(int argc, char *argv[])
{
    int i, j;
    const dim3 blockSize(BLKHSIZE, BLKWSIZE, 1);
    const dim3 gridSize(((DATAHSIZE+BLKHSIZE-1)/BLKHSIZE), ((DATAWSIZE+BLKWSIZE-1)/BLKWSIZE), 1);
// these are just for timing
    clock_t t0, t1, t2;
    double t1sum=0.0;
    double t2sum=0.0;
// overall data set sizes
    const int nr = DATAHSIZE;
    const int nc = DATAWSIZE;
// window dimensions
    const int wr = WSIZE;
    const int wc = WSIZE;
// pointers for data set storage via malloc
    iArray *h_in, *d_in;
    oArray *h_out, *d_out;
// start timing
    t0 = clock();
// allocate storage for data set
    if ((h_in = (iArray *)malloc(((nr+wr)*(nc+wc))*sizeof(int))) == 0) {printf("malloc Fail \n"); exit(1);}
    if ((h_out = (oArray *)malloc((nr*nc)*sizeof(int))) == 0) {printf("malloc Fail \n"); exit(1); }
// synthesize data
    printf("Begin init\n");
    memset(h_in, 0x7F, (nr+wr)*(nc+wc)*sizeof(int));
    memset(h_out, 0x7F, (nr*nc)*sizeof(int));
    for (i=0; i<nc+wc; i+=wc)
      for (j=0; j< nr+wr; j+=wr)
        h_in[i][j] = CHECK_VAL;
    t1 = clock();
    t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
    printf("Init took %f seconds.  Begin compute\n", t1sum);
// allocate GPU device buffers
    hipMalloc((void **) &d_in, (((nr+wr)*(nc+wc))*sizeof(int)));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_out, ((nr*nc)*sizeof(int)));
    cudaCheckErrors("Failed to allocate device buffer2");
// copy data to GPU
    hipMemcpy(d_out, h_out, ((nr*nc)*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");
    hipMemcpy(d_in, h_in, (((nr+wr)*(nc+wc))*sizeof(int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy2 failure");

    cmp_win<<<gridSize,blockSize>>>(d_out, d_in);
    cudaCheckErrors("Kernel launch failure");
// copy output data back to host

    hipMemcpy(h_out, d_out, ((nr*nc)*sizeof(int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy3 failure");
    t2 = clock();
    t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
    printf ("Done. Compute took %f seconds\n", t2sum);
    for (i=0; i < nc; i++)
      for (j=0; j < nr; j++)
        if (h_out[i][j] != CHECK_VAL) {printf("mismatch at %d,%d, was: %d should be: %d\n", i,j,h_out[i][j], CHECK_VAL); return 1;}
    printf("Results pass\n");

    return 0;
}
